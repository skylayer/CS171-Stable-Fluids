#include "hip/hip_runtime.h"
//
// Created by condo on 2024/1/8.
//

#include "render.cuh"
#include "utils.cuh"


__device__ bool intersect(const Eigen::Vector3f &pos, const Eigen::Vector3f &dir, Eigen::Vector3f &hit) {
    const Eigen::Vector3f min      = Eigen::Vector3f(0, 0, 0);
    const Eigen::Vector3f max      = Eigen::Vector3f(1, 1, 1);
    const Eigen::Vector3f inv_dir  = Eigen::Vector3f(1, 1, 1).cwiseQuotient(dir);
    const Eigen::Vector3f t0s      = (min - pos).cwiseProduct(inv_dir);
    const Eigen::Vector3f t1s      = (max - pos).cwiseProduct(inv_dir);
    const Eigen::Vector3f tsmaller = t0s.cwiseMin(t1s);
    const Eigen::Vector3f tbigger  = t0s.cwiseMax(t1s);
    const float           tmin     = tsmaller.maxCoeff();
    const float           tmax     = tbigger.minCoeff();

    if (tmin > tmax) {
        return false;
    }

    hit = pos + tmin * dir;
    return true;
}


__device__ float density(const float *field, const Eigen::Vector3f &pos) {
    if (pos.x() < 0 || pos.x() > 1 || pos.y() < 0 || pos.y() > 1 || pos.z() < 0 || pos.z() > 1) {
        return 0;
    }

    auto coord = (pos.cwiseProduct(Eigen::Vector3f(CELLS_X - 2, CELLS_Y - 2, CELLS_Z - 2)) + Eigen::Vector3f::Constant(1)).eval();
    return lin_interp({coord.x(), coord.y(), coord.z()}, field);
}

__global__ void density_renderer(const Eigen::Matrix3f &view, const Eigen::Vector3f &pos, const float focal, const float **field, float *output) {
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    const Eigen::Vector3f colors[7] = ALL_COLORS;

    if (x < WINDOW_WIDTH && y < WINDOW_HEIGHT) {
        const auto dir = (view * Eigen::Vector3f(x - WINDOW_WIDTH / 2.0f, y - WINDOW_HEIGHT / 2.0f, -focal)).normalized();
        if (Eigen::Vector3f hit; intersect(pos, dir, hit)) {
            float           accumlatedOpacity = 0;
            Eigen::Vector3f color             = Eigen::Vector3f::Zero();
            float           step              = 0.01;
            int             maxIter           = 1000;

            while (accumlatedOpacity < 1 && maxIter--) {
                float maxDensity = 0;
                for (int i = 0; i < NUM_FLUIDS; i++) {
                    const float d = density(field[i], hit);
                    maxDensity    = fmaxf(maxDensity, d);

                    if (d > 0) {
                        const float opacity = 1 - exp(-d * step);
                        const float weight  = opacity * (1 - accumlatedOpacity);

                        color += weight * colors[i];
                        accumlatedOpacity += weight;
                    }
                }

                hit += step * dir;

                if (hit.x() < 0 || hit.x() > 1 || hit.y() < 0 || hit.y() > 1 || hit.z() < 0 || hit.z() > 1) {
                    break;
                }
            }

            output[3 * (y * WINDOW_WIDTH + x) + 0] = color.x();
            output[3 * (y * WINDOW_WIDTH + x) + 1] = color.y();
            output[3 * (y * WINDOW_WIDTH + x) + 2] = color.z();
        }
    }
}

__host__ void render_density(const Eigen::Matrix3f &view, const Eigen::Vector3f &pos, const float focal, const float **field, float3 *output) {
    const dim3 block_size(32, 32);
    const dim3 grid_size(WINDOW_WIDTH / block_size.x + 1, WINDOW_HEIGHT / block_size.y + 1);

    density_renderer<<<grid_size, block_size>>>(view, pos, focal, field, output);
    hipDeviceSynchronize();
}
